
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>

#define CHECK_CUBLAS_ERROR(err) \
if (err != HIPBLAS_STATUS_SUCCESS) \
    {\
        printf("[%s:%d] CUBLAS error %d\n", __FILE__, __LINE__, err);\
        exit(EXIT_FAILURE);\
    }

void mat_mul_cublas(float *A, float *B, float *C,
                    int ROW_A, int COL_A, int COL_B) {

  float *d_A, *d_B, *d_C;
  float *d_trC;
  float one = 1;
  float zero = 0;

  int M = ROW_A;
  int K = COL_A;
  int N = COL_B;


  //if(ROW_A != COL_A || ROW_A != COL_B)
  //{
  //  printf("Support Square Matrix Only!\n");
  // exit(EXIT_FAILURE);
  //}
  /******************** TODO *********************/

  hipblasHandle_t handle;
  hipblasStatus_t status;

  status = hipblasCreate(&handle);
  CHECK_CUBLAS_ERROR(status);

  hipMalloc(&d_A, sizeof(float)*M*K);
  hipMalloc(&d_B, sizeof(float)*K*N);
  hipMalloc(&d_C, sizeof(float)*M*N);
  hipMalloc(&d_trC, sizeof(float)*N*M);


  status = hipblasSetMatrix(M, K, sizeof(float), A, M, d_A, M);
  CHECK_CUBLAS_ERROR(status);

  status = hipblasSetMatrix(K, N, sizeof(float), B, K, d_B, K);
  CHECK_CUBLAS_ERROR(status);

  status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, &one, d_A, K, d_B, N, &zero, d_trC, M);  // Non_Blocking 이다.
  CHECK_CUBLAS_ERROR(status);

  status = hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, &one, d_trC, N, &zero, d_trC, M, d_C, M);
  CHECK_CUBLAS_ERROR(status);

  status = hipblasGetMatrix(M, N, sizeof(float), d_C, M, C, M);
  CHECK_CUBLAS_ERROR(status);

  status = hipblasDestroy(handle);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
